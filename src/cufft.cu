
#include <hip/hip_runtime.h>
#include <complex>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <iostream>

void cufft_example(std::complex<double>* arrHost, size_t n, int batch)
{
  hipfftHandle plan;
  hipfftComplex *dataDevice;
  hipMalloc((void**)&dataDevice, sizeof (hipfftComplex) * n * batch);

  hipMemcpy(dataDevice, (void*)arrHost, n, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: %s, Failed to allocate\n", hipGetErrorString(hipGetLastError()));
    return;
  }

  if (hipfftPlan1d(&plan, n, HIPFFT_C2C, batch) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return;
  }

  /* Note:
   *  Identical pointers to input and output arrays implies in-place transformation
   */

  if (hipfftExecC2C(plan, dataDevice, dataDevice, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
    return;
  }

  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return;
  }

  /*
   *  Divide by number of elements in dataDevice set to get back original data
   */

  hipfftDestroy(plan);
  hipFree(dataDevice);
}
